#include "hip/hip_runtime.h"

#define SIZE 2048

__global__ void VectorAddKernel( float * Vector1, float * Vector2, float * Octput)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  Output[idx] = Vector1[idx] + Vector2[idx];
}
