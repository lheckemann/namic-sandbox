#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */



// A simple macro to divide and round-up
#define DIVUP(A,B) ( (A)%(B) == 0 ? (A)/(B) : ((A) / (B) + 1) )



// We are going to use texture fetching to access the source image.  This allows for effecient
// non-uniform access to image memory and includes hardware bilinear interpolation.  
// The data type, dimension, and read-mode are compile-time set paramters
texture<uchar4, 2, hipReadModeNormalizedFloat> ImageTex;


// Here is the kernel which performs the rotate for every pixel in the output image
__global__ void RotateKernel(uchar4 * out_image, int w, int h, int pitch, float angle)
{
  int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
  int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
   
  if(x < w && y < h)
  {
    // variables to hold the center coordinate of the image.
      // Texture operations use floating point coordinates
    float xf = (float)x - (float)w / 2.0f;
    float yf = (float)y - (float)h / 2.0f;

    // transform to polar coordinates
    float r = sqrtf((float)(xf*xf + yf*yf));
    float theta = atan2f(xf,yf) + 3.14159f / 2.0f;
      
    // compute the new theta by adding the rotation angle
    theta += angle * (3.14159f / 180.0f);

    

    // convert back to cartesian coordinates oriented at 0,0
    yf = r * sinf(theta);   
    yf += (float)h / 2.0f;
    
    xf = -r * cosf(theta);       
    xf += (float)w / 2.0f;  

    // perform the texture lookup to retrieve the new value.
    // It will be returned as a 0 -> 1.0 normalized float
    // because we are doing hardware bilinear interpolation
    float4 out_val_f = tex2D(ImageTex,xf,yf);
    
    // convert back to unsigned char and scale.
    uchar4 out_val;
    out_val.x = (unsigned char)(255.0f * out_val_f.x);
    out_val.y = (unsigned char)(255.0f * out_val_f.y);
    out_val.z = (unsigned char)(255.0f * out_val_f.z);
    out_val.w = 255;
    
    // write the uchar4 value to GMEM
    out_image[__mul24(y,pitch) + x] = out_val;
  }

}

// the Function doRotate is a host function which invokes the kernel to execute the rotate
extern "C" hipError_t RotateImage(hipArray * imageArray, uchar4 * out_image, int width, int height, size_t out_pitch, float angle)
{

  hipError_t c_err;

  // We want to use the hardware bilinear interpolation
  // for improved quality when we modify the image.  
  // This is enabled by setting the runtime filterMode 
  // parameter of the texture.
  ImageTex.filterMode = hipFilterModeLinear;
  ImageTex.addressMode[0] = hipAddressModeClamp;
  ImageTex.addressMode[1] = hipAddressModeClamp;

  // The first step is to bind the hipArray countaining
  // the source image to the texture.
  c_err = hipBindTextureToArray(ImageTex, imageArray);
  if(c_err != hipSuccess) return c_err;
  // we need to create a 2-d thread block and a 2-d grid
  // of blocks.  Lets just make the blocks 16x16.   
  dim3 BlockSz(16,16,1);
  dim3 GridSz(DIVUP(width,BlockSz.x),DIVUP(height,BlockSz.y),1);
  
  // Now launch the kernel to do the rotation
  RotateKernel<<<GridSz,BlockSz>>>(out_image,width,height,(int)out_pitch / 4,angle);
  c_err = hipGetLastError();
  if(c_err != hipSuccess) return c_err;


  c_err = hipDeviceSynchronize();

  // Unbind the texture
  hipUnbindTexture(ImageTex);

      
  return(c_err);
}



texture<unsigned char, 1, hipReadModeElementType> GBR_Tex;

__global__ void convertBGR_U8_to_RGBA_U8_kernel(uchar4 * dest, int width, int height, int dest_pitch_32b)
{
    // compute the X & Y coordinates of the first pixel this thread will process
    int X = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    int Y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
    
    if(X < width && Y < height)
  {
        uchar4 out_val;
                
        int pix_idx = __mul24(Y,__mul24(width,3)) + __mul24(X,3);

    out_val.z = tex1Dfetch(GBR_Tex,pix_idx);
    out_val.y = tex1Dfetch(GBR_Tex,pix_idx+1);
    out_val.x = tex1Dfetch(GBR_Tex,pix_idx+2);
    out_val.w = 255;
       
        // write the result to GMEM
        dest[__mul24(Y,dest_pitch_32b) + X] = out_val;
    }
}



extern "C" hipError_t convertBGRtoRGBA(unsigned char * pGPUtemp,uchar4 * pGPUImage, int width, int height, size_t out_pitch)
{
  hipChannelFormatDesc uchar_desc = hipCreateChannelDesc<unsigned char>();
  hipBindTexture(NULL,GBR_Tex,pGPUtemp,uchar_desc);

  dim3 BlockSz(16,16,1);
  dim3 GridSz(DIVUP(width,BlockSz.x),DIVUP(height,BlockSz.y),1);

  convertBGR_U8_to_RGBA_U8_kernel<<<GridSz,BlockSz>>>(pGPUImage, width, height, (int)out_pitch / 4);

  hipUnbindTexture(GBR_Tex);

  return(hipGetLastError());

}
