#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels

#include "CUDA_renderSlice.h"
#include "CUDA_matrix_math.h"
#include "CUDA_zbuffer_math.h"

#include "vtkType.h"

//#define USE_TIMER
#define BLOCK_DIM2D 8 // this must be set to 4 or more
#define ACC(X,Y,Z) ( ( (Z)*(sizeX)*(sizeY) ) + ( (Y)*(sizeX) ) + (X) )
#define SQR(X) ((X) * (X) )

__constant__ cudaRendererInformation cRenInfo;
__constant__ cudaVolumeInformation cVolInfo;

__constant__ float colorTF[256*3];
__constant__ float alphaTF[256];

template <typename T>
__device__ T CUDAkernel_InterpolateNN(T * sourceData,
				      float posX, 
				      float posY, 
				      float posZ){
  return sourceData[(int)(__float2int_rn(posZ)*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+__float2int_rn(posY)*cVolInfo.VolumeSize.x+__float2int_rn(posX))];
}

template <typename T>
__device__ T CUDAkernel_InterpolateTrilinear(T * sourceData,
					     float posX, 
					     float posY, 
					     float posZ){
  
  
  float fracX=posX-(int)posX;
  float fracY=posY-(int)posY;
  float fracZ=posZ-(int)posZ;
  
  float revX=1-fracX;
  float revY=1-fracY;
  float revZ=1-fracZ;
  
  int base=(int)((int)(posZ)*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+(int)(posY)*cVolInfo.VolumeSize.x+(int)(posX));
  
  return ((T) (revX*(revY*(revZ* (sourceData)[(int)(base)]+
			   fracZ* (sourceData)[(int)(base+cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y)])+
		     fracY*(revZ* (sourceData)[(int)(base+cVolInfo.VolumeSize.x)]+
			    fracZ* (sourceData)[(int)(base+cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+cVolInfo.VolumeSize.x)]))+
	       fracX*(revY*(revZ* (sourceData)[(int)(base+1)]+
			    fracZ* (sourceData)[(int)(base+cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+1)])+
		      fracY*(revZ* (sourceData)[(int)(base+cVolInfo.VolumeSize.x+1)]+
			     fracZ* (sourceData)[(int)(base+cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+cVolInfo.VolumeSize.x+1)])))
	  );
}

template <typename T>
__device__ T CUDAkernel_Interpolate(T * sourceData,
				    float posX, 
				    float posY, 
				    float posZ){
  if(cRenInfo.interpolationMethod == 0){
    return CUDAkernel_InterpolateNN(sourceData, posX, posY, posZ);
  }else if(cRenInfo.interpolationMethod == 1){
    return CUDAkernel_InterpolateTrilinear(sourceData, posX, posY, posZ);
  }
  return 0;
}

__device__ void CUDAkernel_renderSlice_SetRayMapVolumeRendering(long int base, float3* rayMap, float3* clippingPoints, long int index, float* lensMap, int xIndex, int yIndex){
  
  float3 start;
  float3 end;

  start.x=cRenInfo.CameraRayStart.x+
    (float)xIndex/(cRenInfo.ActualResolution.x-1)*cRenInfo.CameraRayStartX.x+
    (float)yIndex/(cRenInfo.ActualResolution.y-1)*cRenInfo.CameraRayStartY.x;
  start.y=cRenInfo.CameraRayStart.y+
    (float)xIndex/(cRenInfo.ActualResolution.x-1)*cRenInfo.CameraRayStartX.y+
    (float)yIndex/(cRenInfo.ActualResolution.y-1)*cRenInfo.CameraRayStartY.y;
  start.z=cRenInfo.CameraRayStart.z+
    (float)xIndex/(cRenInfo.ActualResolution.x-1)*cRenInfo.CameraRayStartX.z+
    (float)yIndex/(cRenInfo.ActualResolution.y-1)*cRenInfo.CameraRayStartY.z;

  end.x=cRenInfo.CameraRayEnd.x+
    (float)xIndex/(cRenInfo.ActualResolution.x-1)*cRenInfo.CameraRayEndX.x+
    (float)yIndex/(cRenInfo.ActualResolution.y-1)*cRenInfo.CameraRayEndY.x;
  end.y=cRenInfo.CameraRayEnd.y+
    (float)xIndex/(cRenInfo.ActualResolution.x-1)*cRenInfo.CameraRayEndX.y+
    (float)yIndex/(cRenInfo.ActualResolution.y-1)*cRenInfo.CameraRayEndY.y;
  end.z=cRenInfo.CameraRayEnd.z+
    (float)xIndex/(cRenInfo.ActualResolution.x-1)*cRenInfo.CameraRayEndX.z+
    (float)yIndex/(cRenInfo.ActualResolution.y-1)*cRenInfo.CameraRayEndY.z;

  rayMap[base*2].x=start.x;
  rayMap[base*2].y=start.y;
  rayMap[base*2].z=start.z;

  rayMap[base*2+1].x=end.x-start.x;
  rayMap[base*2+1].y=end.y-start.y;
  rayMap[base*2+1].z=end.z-start.z;

  rayMap[base*2]=MatMul(cVolInfo.Transform, rayMap[base*2]);
  rayMap[base*2+1]=MatMul(cVolInfo.Transform, rayMap[base*2+1], 0.0f);

  clippingPoints[base*2].x=rayMap[base*2].x;
  clippingPoints[base*2].y=rayMap[base*2].y;
  clippingPoints[base*2].z=rayMap[base*2].z;

  clippingPoints[base*2+1].x=rayMap[base*2].x+rayMap[base*2+1].x;
  clippingPoints[base*2+1].y=rayMap[base*2].y+rayMap[base*2+1].y;
  clippingPoints[base*2+1].z=rayMap[base*2].z+rayMap[base*2+1].z;

  float getmax = fabs(rayMap[base*2+1].x);
  if(fabs(rayMap[base*2+1].y)>getmax) getmax = fabs(rayMap[base*2+1].y);
  if(fabs(rayMap[base*2+1].z)>getmax) getmax = fabs(rayMap[base*2+1].z);
  
  if(getmax!=0){
    float temp= 1.0f/getmax;
    rayMap[base*2+1].x*=temp;
    rayMap[base*2+1].y*=temp;
    rayMap[base*2+1].z*=temp;
  }
}

__device__ void CUDAkernel_renderSlice_CalculateMinmax(long int tempacc, float3* rayMap, float2* minmax, int xindex, int yindex){

  float3 zVec;
  float3 normalVec;
  float3 newOrigin;

  zVec.x=0;
  zVec.y=0;
  zVec.z=1;

  normalVec=MatMul(cVolInfo.SliceMatrix, zVec, 0.0f);
  newOrigin=MatMul(cVolInfo.SliceMatrix, make_float3(0,0,0), 1.0f);

  float3 transformedOrigin;
  float3 transformedVector;

  transformedVector=MatMul(cVolInfo.OrientationMatrix, rayMap[2*tempacc+1], 0.0f);
  transformedOrigin=MatMul(cVolInfo.OrientationMatrix, rayMap[2*tempacc], 1.0f);
  
  float length=sqrt(normalVec.x*normalVec.x+
		    normalVec.y*normalVec.y+
		    normalVec.z*normalVec.z);
  
  normalVec.x/=length;
  normalVec.y/=length;
  normalVec.z/=length;
  
  float3 relPos;
  
  relPos.x=newOrigin.x-transformedOrigin.x;
  relPos.y=newOrigin.y-transformedOrigin.y;
  relPos.z=newOrigin.z-transformedOrigin.z;

  float unit=(relPos.x*normalVec.x+
	      relPos.y*normalVec.y+
	      relPos.z*normalVec.z)/(
				     transformedVector.x*normalVec.x+
				     transformedVector.y*normalVec.y+
				     transformedVector.z*normalVec.z);

  
  minmax[tempacc].x=unit;
  minmax[tempacc].y=unit;
}

template <typename T>
__global__ void CUDAkernel_renderSlice_calculateShadeField()
{
  int xIndex = (blockDim.x*blockIdx.x + threadIdx.x) % (int)cVolInfo.VolumeSize.x;
  int yIndex = (blockDim.x*blockIdx.x + threadIdx.x) / (int)cVolInfo.VolumeSize.x;
  int zIndex = blockDim.y*blockIdx.y+ threadIdx.y;

  long int index = (xIndex+yIndex*cVolInfo.VolumeSize.x+zIndex*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y);

  float3 tempShade;

  if(xIndex>0 && xIndex < cVolInfo.VolumeSize.x-1 && yIndex>0 && yIndex < cVolInfo.VolumeSize.y-1 && zIndex>0 && zIndex < cVolInfo.VolumeSize.z-1){
    tempShade.x = (float)((T*)cVolInfo.SourceData)[(int)(__float2int_rn(zIndex)*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+__float2int_rn(yIndex)*cVolInfo.VolumeSize.x+__float2int_rn(xIndex+1))];
    tempShade.y = (float)((T*)cVolInfo.SourceData)[(int)(__float2int_rn(zIndex)*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+__float2int_rn(yIndex+1)*cVolInfo.VolumeSize.x+__float2int_rn(xIndex))];
    tempShade.z = (float)((T*)cVolInfo.SourceData)[(int)(__float2int_rn(zIndex+1)*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+__float2int_rn(yIndex)*cVolInfo.VolumeSize.x+__float2int_rn(xIndex))];

    tempShade.x-=(float)((T*)cVolInfo.SourceData)[(int)(__float2int_rn(zIndex)*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+__float2int_rn(yIndex)*cVolInfo.VolumeSize.x+__float2int_rn(xIndex-1))];
    tempShade.y-=(float)((T*)cVolInfo.SourceData)[(int)(__float2int_rn(zIndex)*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+__float2int_rn(yIndex-1)*cVolInfo.VolumeSize.x+__float2int_rn(xIndex))];
    tempShade.z-=(float)((T*)cVolInfo.SourceData)[(int)(__float2int_rn(zIndex-1)*cVolInfo.VolumeSize.x*cVolInfo.VolumeSize.y+__float2int_rn(yIndex)*cVolInfo.VolumeSize.x+__float2int_rn(xIndex))];
  }else if((xIndex==0 || xIndex == cVolInfo.VolumeSize.x-1) && (yIndex==0 || yIndex == cVolInfo.VolumeSize.y-1) && (zIndex==0 || zIndex == cVolInfo.VolumeSize.z-1)){
    
    tempShade.x=0;
    tempShade.y=0;
    tempShade.z=0;
  }else{
    index=-1;
  }
  
  if(index!=-1){
    cVolInfo.shadeField[index].x=tempShade.x;
    cVolInfo.shadeField[index].y=tempShade.y;
    cVolInfo.shadeField[index].z=tempShade.z;
  }
}

template <typename T>
__global__ void CUDAkernel_renderSlice_doRendering()
{
  
  int xIndex = blockDim.x *blockIdx.x + threadIdx.x;
  int yIndex = blockDim.y *blockIdx.y + threadIdx.y;
  
  __shared__ float2 s_minmaxTrace[BLOCK_DIM2D*BLOCK_DIM2D];
  __shared__ float3 s_rayMap[BLOCK_DIM2D*BLOCK_DIM2D*2];
  __shared__ float3 s_clippingPoints[BLOCK_DIM2D*BLOCK_DIM2D*2];
  
  int tempacc=threadIdx.x+threadIdx.y*BLOCK_DIM2D;
  
  __syncthreads();
  
  long int index = (xIndex+yIndex*cRenInfo.ActualResolution.x)*4;
  
  if(xIndex<cRenInfo.ActualResolution.x && yIndex <cRenInfo.ActualResolution.y){  
    CUDAkernel_renderSlice_SetRayMapVolumeRendering(tempacc, s_rayMap, s_clippingPoints, index, cRenInfo.LensMap, xIndex, yIndex);
        
    CUDAkernel_renderSlice_CalculateMinmax(tempacc, s_rayMap, s_minmaxTrace, xIndex, yIndex);
    
    float tempx, tempy, tempz;
    float temp;
    float r=0;
    float g=0;
    float b=0;
    float alpha=0;

    float zBuffer;
    float newZBuffer = cRenInfo.ZBuffer[cRenInfo.ActualResolution.x-1-xIndex+yIndex*cRenInfo.ActualResolution.x];
    
    tempx = ( s_rayMap[tempacc*2].x+((int)s_minmaxTrace[tempacc].x)*s_rayMap[tempacc*2+1].x);
    tempy = ( s_rayMap[tempacc*2].y+((int)s_minmaxTrace[tempacc].x)*s_rayMap[tempacc*2+1].y);
    tempz = ( s_rayMap[tempacc*2].z+((int)s_minmaxTrace[tempacc].x)*s_rayMap[tempacc*2+1].z);
    
    if(tempx >= cVolInfo.minROI.x+1 && tempx < cVolInfo.maxROI.x-1 && tempy >= cVolInfo.minROI.y+1 && tempy < cVolInfo.maxROI.y-1 && tempz >= cVolInfo.minROI.z+1 && tempz < cVolInfo.maxROI.z-1){
      
      CalculateZBuffer(cRenInfo, &zBuffer, s_clippingPoints[tempacc*2].x, s_clippingPoints[tempacc*2+1].x, tempx);

      temp=CUDAkernel_Interpolate((T*)cVolInfo.SourceData, tempx, tempy, tempz);

      if(temp >=cVolInfo.MinThreshold && temp <= cVolInfo.MaxThreshold){
	
	if(zBuffer < cRenInfo.ZBuffer[cRenInfo.ActualResolution.x-1-xIndex+yIndex*cRenInfo.ActualResolution.x]){
	  temp=(float)(temp-cVolInfo.TypeRange[0])/(float)(cVolInfo.TypeRange[1]-cVolInfo.TypeRange[0])*(cVolInfo.FunctionSize-1);
	  
	  r=colorTF[(int)temp*3]* 255.0;
	  g=colorTF[(int)temp*3+1]* 255.0;
	  b=colorTF[(int)temp*3+2]* 255.0;
	  alpha=255.0;
	  
	  newZBuffer = zBuffer;
	}
      }  
    }
    
    cRenInfo.ZBuffer[cRenInfo.ActualResolution.x-1-xIndex+yIndex*cRenInfo.ActualResolution.x]=newZBuffer;
    cRenInfo.OutputImage[(int)(xIndex+yIndex*cRenInfo.ActualResolution.x)]=make_uchar4(r, g, b, alpha);
  }
  
}

void CUDArenderSlice_doRender(cudaRendererInformation& renInfo, cudaVolumeInformation& volInfo)
{
  // setup execution parameters

  int blockX=(((int)renInfo.ActualResolution.x-1)/ BLOCK_DIM2D) + 1;
  int blockY=(((int)renInfo.ActualResolution.y-1)/ BLOCK_DIM2D) + 1;
  
  // setup execution parameters
  
  dim3 grid(blockX, blockY, 1);
  dim3 threads(BLOCK_DIM2D, BLOCK_DIM2D, 1);

  blockX=((int)(volInfo.VolumeSize.x*volInfo.VolumeSize.y-1)/ BLOCK_DIM2D) + 1;
  blockY=((int)(volInfo.VolumeSize.z-1)/ BLOCK_DIM2D) + 1;

  dim3 grid2(blockX, blockY, 1);
  dim3 threads2(BLOCK_DIM2D, BLOCK_DIM2D, 1);

  // copy host memory to device

  prepareShadeFieldSlice(renInfo, volInfo);

  CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(cRenInfo), &renInfo, sizeof(cudaRendererInformation)));
  CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(cVolInfo), &volInfo, sizeof(cudaVolumeInformation)));

  CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(colorTF), volInfo.ColorTransferFunction, sizeof(float)*256*3));
  CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(alphaTF), volInfo.AlphaTransferFunction, sizeof(float)*256));

  //execute the kernel
  if(renInfo.rayCastingMethod==3){ //if CompositeShaded
#define CALCULATE_SHADE_FIELD_SLICE(ID, TYPE)					\
    (ID==volInfo.InputDataType)						\
      CUDAkernel_renderSlice_calculateShadeField<TYPE><<<grid2, threads2>>>()

    if CALCULATE_SHADE_FIELD_SLICE(VTK_CHAR, char);
    else if CALCULATE_SHADE_FIELD_SLICE(VTK_CHAR, char);
    else if CALCULATE_SHADE_FIELD_SLICE(VTK_UNSIGNED_CHAR, unsigned char);
    else if CALCULATE_SHADE_FIELD_SLICE(VTK_SHORT, short);
    else if CALCULATE_SHADE_FIELD_SLICE(VTK_UNSIGNED_SHORT, unsigned short);
    else if CALCULATE_SHADE_FIELD_SLICE(VTK_INT, int);
    else if CALCULATE_SHADE_FIELD_SLICE(VTK_FLOAT, float);
    
    //CUDAkernel_renderSlice_calculateShadeField<unsigned char><<<grid2, threads2>>>();
  }

#define CALL_KERNEL_DO_RENDER(ID, TYPE)					\
  (ID==volInfo.InputDataType)						\
    CUDAkernel_renderSlice_doRendering<TYPE><<<grid, threads>>>()
  
  if CALL_KERNEL_DO_RENDER(VTK_CHAR, char);
  else if CALL_KERNEL_DO_RENDER(VTK_CHAR, char);
  else if CALL_KERNEL_DO_RENDER(VTK_UNSIGNED_CHAR, unsigned char);
  else if CALL_KERNEL_DO_RENDER(VTK_SHORT, short);
  else if CALL_KERNEL_DO_RENDER(VTK_UNSIGNED_SHORT, unsigned short);
  else if CALL_KERNEL_DO_RENDER(VTK_INT, int);
  else if CALL_KERNEL_DO_RENDER(VTK_FLOAT, float);

  deleteShadeFieldSlice(renInfo, volInfo);

  CUT_CHECK_ERROR("Kernel execution failed");

  return;
}

void prepareShadeFieldSlice(cudaRendererInformation& renInfo, cudaVolumeInformation& volInfo){
  CUDA_SAFE_CALL(hipMalloc((void**)&volInfo.shadeField, (int)(volInfo.VolumeSize.x*volInfo.VolumeSize.y*volInfo.VolumeSize.z*sizeof(float3))));
}

void deleteShadeFieldSlice(cudaRendererInformation& renInfo, cudaVolumeInformation& volInfo){
  CUDA_SAFE_CALL( hipFree(volInfo.shadeField));
}


