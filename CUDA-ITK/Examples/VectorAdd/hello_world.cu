#include "hip/hip_runtime.h"





#include <stdio.h>

#define SIZE 2050
#define DIVUP(a,b) (a % b) == 0 ? (a / b) : (a / b) + 1


__global__ void VectorAddKernel(float * Vector1, float * Vector2, float * Output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < size)
        Output[idx] = Vector1[idx] + Vector2[idx];
}


int main()
{
    float HostVector1[SIZE];
    float HostVector2[SIZE];
    float HostOutputVector[SIZE];

    for(int i=0;i<SIZE;i++)
    {
        HostVector1[i] = i;
        HostVector2[i] = i;
    }

    float * GPUVector1;
    float * GPUVector2;
    float * GPUOutputVector;

    hipError_t err;
    err = hipMalloc((void**)&GPUVector1,SIZE*sizeof(float));
    err = hipMalloc((void**)&GPUVector2,SIZE*sizeof(float));
    err = hipMalloc((void**)&GPUOutputVector,SIZE*sizeof(float));

    err = hipMemcpy(GPUVector1,HostVector1,SIZE*sizeof(float),hipMemcpyHostToDevice);
    err = hipMemcpy(GPUVector2,HostVector2,SIZE*sizeof(float),hipMemcpyHostToDevice);

    dim3 BlockDim(64,1,1);
    dim3 GridDim(DIVUP(SIZE,BlockDim.x),1,1);

    VectorAddKernel<<<GridDim,BlockDim>>>(GPUVector1,GPUVector2,GPUOutputVector,SIZE);
    
    // Do other stuff...

    hipDeviceSynchronize();

    err = hipMemcpy(HostOutputVector,GPUOutputVector,SIZE*sizeof(float),hipMemcpyDeviceToHost);
    
    err = hipFree(GPUVector1);
    err = hipFree(GPUVector2);
    err = hipFree(GPUOutputVector);

    for(int i=0;i<SIZE; i++)
        printf("%8.3f\n",HostOutputVector[i]);



}
