#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */


// A simple macro to divide and round-up
#define DIVUP(A,B) ( (A)%(B) == 0 ? (A)/(B) : ((A) / (B) + 1) )

// macro to clamp to min & max value:
#define CLAMP(A,B,C) ( (A) < (B) ? (B) : (A) > (C) ? (C) : (A) )

// Declare constant memory for our convolution filter
__constant__ float FilterCoeff[9];

// Here's the CPU data for our filter which we'll copy into the constant memory
float FilterCoeff_CPU[9] = { -1.0f, -2.0f, -1.0f,
               -2.0f, 12.0f, -2.0f,
               -1.0f, -2.0f, -1.0f};



#define BLOCK_W 16
#define BLOCK_H 16
#define TILE_W (BLOCK_W * 4 + 16)  // need 8 pixel aprons because we are doing 64-bit reads)
#define TILE_H (BLOCK_H + 2)


__global__ void Convolution3x3Kernel(uchar4 * in_image, uchar4 * out_image, int width, int height, int pitch32)
{

  // This is a pointer to the shared memory we use for the image tile
  __shared__ unsigned char smem[TILE_W*TILE_H];
  int2 * smem_64bit = (int2*)smem;

  // Compute the thread index overall
  int X = __mul24(blockDim.x,blockIdx.x) + threadIdx.x;  
  int Y = __mul24(blockDim.y,blockIdx.y) + threadIdx.y;
  
  // variables to hold the addresses we are going to write & read in GMEM & SMEM
  int smem_idx;
  int gmem_idx;

  // Perform 64-bit reads, so only some of the threads need to participate.  
  // we'll cast the source image to a 64-bit data type (e.g. int2) and them read into a the smem with the same
  // 64-bit casting
  
  if(threadIdx.x < BLOCK_W / 2 + 2)
  {
    int row = CLAMP(Y-1,0,height-1);
    int col = __mul24(blockDim.x>>1,blockIdx.x) + threadIdx.x - 1;  // reading 8 bytes / thread, but only 1/2 of the block is used so divide blockDim / 2
    col = CLAMP(col,0,width>>3-1);
    gmem_idx = __mul24(row,pitch32>>1) + col;            // must divide the pitch32 & X by 2 since it's a 64-bit address
    smem_idx = __mul24(threadIdx.y,TILE_W/8) + threadIdx.x;        // the index into the 64-bit casted smem
  
    // Perform the read
    smem_64bit[smem_idx] = ((int2*)in_image)[gmem_idx];

    // two rows need to read again for the lower apron rows
    if(threadIdx.y < 2)
    {
      row = CLAMP(Y + BLOCK_H - 1,0,height-1);      
      gmem_idx = __mul24(row,pitch32>>1) + col;  
      smem_idx += TILE_W/8 * BLOCK_H;
      smem_64bit[smem_idx] = ((int2*)in_image)[gmem_idx];
    }
  }
    

  // Now we need to wait until all threads in this block have finished reading
  // their respective pixels into SMEM
  __syncthreads();
  if(X < width && Y < height) 
  {
    // Now for the convolutions.  
    uchar4 out;  // variable to store our output
    unsigned int f_idx = 0;  // index into the filter coefficients
    float sum;  

    // compute the location of the pixel in smem we are going to start to process.  
    // this will be the pixel to the top-left of the active pixel
    smem_idx = __mul24(threadIdx.y,TILE_W) + (threadIdx.x<<2) +7;

    // now do the math.  We'll use floating point since it's fastest
    // Pixel 1
    sum = FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+2];
    smem_idx += TILE_W;
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+2];
    smem_idx += TILE_W;
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx] * (float)smem[smem_idx+2];
    out.x = (unsigned char)CLAMP(sum,0,255.0f);

    // Pixel 2
    f_idx = 0;
    smem_idx = __mul24(threadIdx.y,TILE_W) + (threadIdx.x<<2) +8;
    sum = FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+2];
    smem_idx += TILE_W;
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+2];
    smem_idx += TILE_W;
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx] * (float)smem[smem_idx+2];
    out.y = (unsigned char)CLAMP(sum,0,255.0f);

    // Pixel 3
    f_idx = 0;
    smem_idx = __mul24(threadIdx.y,TILE_W) + (threadIdx.x<<2) + 9;
    sum = FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+2];
    smem_idx += TILE_W;
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+2];
    smem_idx += TILE_W;
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx] * (float)smem[smem_idx+2];
    out.z = (unsigned char)CLAMP(sum,0,255.0f);

    // Pixel 4
    f_idx = 0;
    smem_idx = __mul24(threadIdx.y,TILE_W) + (threadIdx.x<<2) +10;
    sum = FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+2];
    smem_idx += TILE_W;
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+2];
    smem_idx += TILE_W;
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx];
    sum += FilterCoeff[f_idx++] * (float)smem[smem_idx+1];
    sum += FilterCoeff[f_idx] * (float)smem[smem_idx+2];
    out.w = (unsigned char)CLAMP(sum,0,255.0f);

    // **DEBUG**
    // The following lines of code are useful for debugging 
    // They simply copy the value of the primary pixels back to the output rather
    // than performing the convolution.  Thus, it's easy to tell if the GMEM is loaded
    // into SMEM properly. It's also useful to shift the offsets to insure the aprons are correct
    //out.x = smem[(threadIdx.y+1)*TILE_W + threadIdx.x*4+7];
    //out.y = smem[(threadIdx.y+1)*TILE_W + threadIdx.x*4+8];
    //out.z = smem[(threadIdx.y+1)*TILE_W + threadIdx.x*4+9];
    //out.w = smem[(threadIdx.y+1)*TILE_W + threadIdx.x*4+10];

    
    // Finally, write out the result!    
    gmem_idx = __mul24(Y,pitch32) + X;
    out_image[gmem_idx] = out;
  }
}



// Function to preform an arbitrary 3x3 convolution on a grayscale 8-bit image
extern "C" hipError_t Convolution3x3(unsigned char * src, unsigned char * dest, int width, int height, size_t pitch)
{
  hipError_t cerr;

  // First, copy data for the filter coefficients into constant memory
  cerr = hipMemcpyToSymbol(HIP_SYMBOL(FilterCoeff),FilterCoeff_CPU,9*sizeof(float));
  if(cerr != hipSuccess) return(cerr);

  // Compute our block & grid dimensions
  dim3 BlockSz(BLOCK_W,BLOCK_H,1);
  dim3 GridSz(DIVUP(width,BlockSz.x*4),DIVUP(height,BlockSz.y),1);
  
  
  Convolution3x3Kernel<<<GridSz,BlockSz>>>((uchar4*)src, (uchar4*)dest,
                            width, height, (int)pitch/4);

  // Wait for this function to complete before we return
  cerr = hipDeviceSynchronize();

  return(cerr);  

}
