#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */


// A simple macro to divide and round-up
#define DIVUP(A,B) ( (A)%(B) == 0 ? (A)/(B) : ((A) / (B) + 1) )

// macro to clamp to min & max value:
#define CLAMP(A,B,C) ( (A) < (B) ? (B) : (A) > (C) ? (C) : (A) )


// Here is the kernel which performs the rotate for every pixel in the output image
__global__ void BrightnessContrastKernel(uchar4 * image, int w, int h, int pitch, float brightness, float contrast)
{
  // compute the x & y coordinates in the image the current pixel must process.  

  // the image data is passed in as a uchar4, so this x coordinate 
  // points 4-pixel groups
  int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x; 
  int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
   
  if(x < (w>>2) && y < h)    // w/4 because we are processing 4 pixels per thread
  {

    float temp;

    // read in the value to modify
    uchar4 pixel = image[__umul24(y,pitch>>2) + x];

    // Pixel 1 
    temp = ((float)pixel.x + brightness) * contrast;
    pixel.x = CLAMP(temp,0.0f,255.0f);

    // Pixel 2 
    temp = ((float)pixel.y + brightness) * contrast;
    pixel.y = CLAMP(temp,0.0f,255.0f);

    // Pixel 3
    temp = ((float)pixel.z + brightness) * contrast;
    pixel.z = CLAMP(temp,0.0f,255.0f);

    // Pixel 4
    temp = ((float)pixel.w + brightness) * contrast;
    pixel.w = CLAMP(temp,0.0f,255.0f);

    // write the new pixel value back to the image data
    image[__umul24(y,pitch>>2) + x] = pixel;
  }

}

// This function will adjust the brightness and contrast an image
extern "C" hipError_t AdjustBrightnessContrast(unsigned char * image, int width, int height, 
                          int pitch, float brightness_adjust, float contrast_adjust)
{
  
  // we need to create a 2-d thread block and a 2-d grid
  // of blocks.  Lets just make the blocks 16x14. 
  // Process four pixels / thread
  dim3 BlockSz(16,14,1);
  dim3 GridSz(DIVUP(width,BlockSz.x*4),DIVUP(height,BlockSz.y),1);

  // Now launch the kernel to do the adjustment
  BrightnessContrastKernel<<<GridSz,BlockSz>>>((uchar4*)image,width,height,pitch,brightness_adjust,contrast_adjust);
  
  // just wait here for the kernel to complete
  hipError_t err = hipDeviceSynchronize();  

  return(err);
}


