#include "hip/hip_runtime.h"
/*=========================================================================

  Program:   Insight Segmentation & Registration Toolkit
  Module:    $RCSfile: itkImage.h,v $
  Language:  C++
  Date:      $Date: 2009-02-05 19:04:56 $
  Version:   $Revision: 1.150 $

  Copyright (c) Insight Software Consortium. All rights reserved.
  See ITKCopyright.txt or http://www.itk.org/HTML/Copyright.htm for details.

     This software is distributed WITHOUT ANY WARRANTY; without even
     the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
     PURPOSE.  See the above copyright notices for more information.

=========================================================================*/

#include "Example2.h"


/**
**************************************************************************
*  Memory allocator, returns aligned format frame with 32bpp float pixels.
*
* \param width			[IN] - Width of image buffer to be allocated
* \param height			[IN] - Height of image buffer to be allocated
* \param pStepBytes		[OUT] - Step between two sequential rows
*  
* \return Pointer to the created plane
*/
float *MallocPlaneFloat(int width, int height, int *pStepBytes)
{
	float *ptr;
	*pStepBytes = ((int)ceil((width*sizeof(float))/16.0f))*16;
//#ifdef __ALLOW_ALIGNED_MEMORY_MANAGEMENT
//	ptr = (float *)_aligned_malloc(*pStepBytes * height, 16);
//#else
	ptr = (float *)malloc(*pStepBytes * height);
//#endif
	*pStepBytes = *pStepBytes / sizeof(float);
	return ptr;
}

/**
**************************************************************************
*  Copies byte plane to float plane
*
* \param ImgSrc				[IN] - Source byte plane
* \param StrideB			[IN] - Source plane stride
* \param ImgDst				[OUT] - Destination float plane
* \param StrideF			[IN] - Destination plane stride
* \param Size				[IN] - Size of area to copy
*  
* \return None
*/
void CopyByte2Float(byte *ImgSrc, int StrideB, float *ImgDst, int StrideF, ROI Size)
{
	for (int i=0; i<Size.height; i++)
	{
		for (int j=0; j<Size.width; j++)
		{
			ImgDst[i*StrideF+j] = (float)ImgSrc[i*StrideB+j];
		}
	}
}


/**
**************************************************************************
*  Float round to nearest value
*
* \param num			[IN] - Float value to round
*  
* \return The closest to the input float integer value
*/
float round_f(float num) 
{
	float NumAbs = fabs(num);
	int NumAbsI = (int)(NumAbs + 0.5f);
	float sign = num > 0 ? 1.0f : -1.0f;
	return sign * NumAbsI;
}


/**
**************************************************************************
*  The routine clamps the input value to integer byte range [0, 255]
*
* \param x			[IN] - Input value
*  
* \return Pointer to the created plane
*/
int clamp_0_255(int x)
{
	return (x < 0) ? 0 : ( (x > 255) ? 255 : x );
}


/**
**************************************************************************
*  Copies float plane to byte plane (with clamp)
*
* \param ImgSrc				[IN] - Source float plane
* \param StrideF			[IN] - Source plane stride
* \param ImgDst				[OUT] - Destination byte plane
* \param StrideB			[IN] - Destination plane stride
* \param Size				[IN] - Size of area to copy
*  
* \return None
*/
void CopyFloat2Byte(float *ImgSrc, int StrideF, byte *ImgDst, int StrideB, ROI Size)
{
	for (int i=0; i<Size.height; i++)
	{
		for (int j=0; j<Size.width; j++)
		{
			ImgDst[i*StrideB+j] = (byte)clamp_0_255((int)(round_f(ImgSrc[i*StrideF+j])));
		}
	}
}


/**
**************************************************************************
*  Memory deallocator, deletes aligned format frame.
*
* \param ptr			[IN] - Pointer to the plane
*  
* \return None
*/
void FreePlane(void *ptr)
{
//#ifdef __ALLOW_ALIGNED_MEMORY_MANAGEMENT
//	if (ptr) 
//	{
//		_aligned_free(ptr);
//	}
//#else
	if (ptr) 
	{
		free(ptr);
	}
//#endif
}



void CopyITKImageToCUDA(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size)
{
	//prepare channel format descriptor for passing texture into kernels
	hipChannelFormatDesc floattex = hipCreateChannelDesc<float>();

	//allocate device memory
	hipArray *Src;
	float *Dst;
	size_t DstStride;
	cutilSafeCall(hipMallocArray(&Src, &floattex, Size.width, Size.height));
	cutilSafeCall(hipMallocPitch((void **)(&Dst), &DstStride, Size.width * sizeof(float), Size.height));
	DstStride /= sizeof(float);

	//convert source image to float representation
	int ImgSrcFStride;
	float *ImgSrcF = MallocPlaneFloat(Size.width, Size.height, &ImgSrcFStride);
	CopyByte2Float(ImgSrc, Stride, ImgSrcF, ImgSrcFStride, Size);

	//copy from host memory to device
	cutilSafeCall(hipMemcpy2DToArray(Src, 0, 0,
									   ImgSrcF, ImgSrcFStride * sizeof(float), 
									   Size.width * sizeof(float), Size.height,
									   hipMemcpyHostToDevice) );


	//copy quantized image block to host
	cutilSafeCall(hipMemcpy2D(ImgSrcF, ImgSrcFStride * sizeof(float), 
								Dst, DstStride * sizeof(float), 
								Size.width * sizeof(float), Size.height,
								hipMemcpyDeviceToHost) );

	//convert image back to byte representation
	CopyFloat2Byte(ImgSrcF, ImgSrcFStride, ImgDst, Stride, Size);

	//clean up memory
	cutilSafeCall(hipFreeArray(Src));
	cutilSafeCall(hipFree(Dst));
	FreePlane(ImgSrcF);

}
