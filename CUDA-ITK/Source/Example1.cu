
#include <hip/hip_runtime.h>
#include <stdio.h>

//
//  link to cudart.lib
//
//  This file should be compiled by the "nvcc" compiler.
//

#define SIZE 2048
#define DIVUP(a,b) ( a % b ) == 0 ? (a/b): (a/b) + 1;

__global__ void VectorAddKernel( float * Vector1, float * Vector2, float * Output, int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if( idx < size )
    {
    Output[idx] = Vector1[idx] + Vector2[idx];
    }
}

int main()
{

  float HostVector1[SIZE];
  float HostVector2[SIZE];
  float HostOutputVector[SIZE];


  for( int i=0; i<SIZE; i++)
    {
    HostVector1[i] = i;
    HostVector2[i] = i;
    }

  float * GPUVector1;
  float * GPUVector2;
  float * GPUOutputVector;

  hipError_t err;

  unsigned int totalSize = SIZE* sizeof(float);

  err = hipMalloc((void**)&GPUVector1, totalSize );
  err = hipMalloc((void**)&GPUVector2, totalSize );
  err = hipMalloc((void**)&GPUOutputVector, totalSize );

  hipMemcpy(GPUVector1, HostVector1, totalSize , hipMemcpyHostToDevice);
  hipMemcpy(GPUVector2, HostVector2, totalSize , hipMemcpyHostToDevice);

  //
  // Define here the strategy for defining the distribution of the problem
  //


  // Size of the data block that will be passed to each one of the streaming
  // multi-processors.
  dim3 BlockDim(128,1,1);

  // 
  // Size of the grid of multi-processors that will be used for processing
  // the total amount of data.
  //
  int numberOfProcessors = DIVUP(SIZE, BlockDim.x);
  dim3 GridDim( numberOfProcessors, 1 , 1 );
  // 17 blocks = 2050 / 128



  //
  // This call is asynchronous.
  //
  // Kernels have a timeout of 5 seconds... if the kernel runs for more than 5 seconds
  // The operating system (Microsoft Windows) will consider that the display crashed.
  //
  VectorAddKernel<<<GridDim,BlockDim>>>(GPUVector1,GPUVector2,GPUOutputVector,SIZE);


  //
  // Do other stuff here...
  // 


  //
  // This call will wait until the GPU is done
  // cudaThreadSynchronize();
  //

  err = hipMemcpy( HostOutputVector, GPUOutputVector, totalSize, hipMemcpyDeviceToHost);

  err = hipFree( GPUVector1 );  
  err = hipFree( GPUVector2 );  
  err = hipFree( GPUOutputVector );  

  for(int i=0; i<SIZE; i++)
    {
    printf("%8.3f\n",HostOutputVector[i]);
    }

  if( err )
    {
    printf("err %d", err );
    }
}
