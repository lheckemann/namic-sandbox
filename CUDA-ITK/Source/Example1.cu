#include "hip/hip_runtime.h"
//
//  link to cudart.lib
//
//  This file should be compiled by the "nvcc" compiler.
//

#define SIZE 2048

__global__ void VectorAddKernel( float * Vector1, float * Vector2, float * Octput)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  Output[idx] = Vector1[idx] + Vector2[idx];
}

int main()
{

  float HostVector1[SIZE];
  float HostVector2[SIZE];
  float HostOutputVector[SIZE];


  for( int i=0; i<SIZE; i++)
    {
    HostVector1[i] = i;
    HostVector2[i] = i;
    }

  float * GPUVector1;
  float * GPUVector2;
  float * GPUOutputVector;

  hipError_t err;

  unsigned int totalSize = SIZE* sizeof(float);

  err = hipMalloc((void**)&GPUVector1, totalSize );
  err = hipMalloc((void**)&GPUVector2, totalSize );
  err = hipMalloc((void**)&GPUOutputVector, totalSize );

  hipMemcpy(GPUVector1, HostVector1, totalSize , hipMemcpyHostToDevice);
  hipMemcpy(GPUVector2, HostVector2, totalSize , hipMemcpyHostToDevice);


  dim3 BlockDim(128,1,1);
  dim3 GidDim(DIVUP(SIZE, BlockDim.x),1,1);

  VectorAddKernel<<<GridDim,BlockDim>>(GPUVector1,GPUVector2,GPUOutputVector,SIZE);

  err = cudaMemory( HostOutputVector, GPUOutputVector, 

  err = hipFree( GPUVector1 );  
  err = hipFree( GPUVector2 );  
  err = hipFree( GPUOutputVector );  

  for(int i=0; i<SIZE; i++)
    {
    printf("%8.3f\n",HostOutputVector[i]);
    }
}
