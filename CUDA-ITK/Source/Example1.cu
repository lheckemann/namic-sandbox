#include "hip/hip_runtime.h"

#define SIZE 2048

__global__ void VectorAddKernel( float * Vector1, float * Vector2, float * Octput)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  Output[idx] = Vector1[idx] + Vector2[idx];
}

int main()
{

  float HostVector1[SIZE];
  float HostVector2[SIZE];
  float HostOutputVector[SIZE];


  for( int i=0; i<SIZE; i++)
    {
    HostVector1[i] = i;
    HostVector2[i] = i;
    }

  float * GPUVector1;
  float * GPUVector2;
  float * GPUOutputVector;

  hipError_t err;

  err = hipMalloc((void**)&GPUVector1, SIZE*sizeof(float));
  err = hipMalloc((void**)&GPUVector2, SIZE*sizeof(float));

}
